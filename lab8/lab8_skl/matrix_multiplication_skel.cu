

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>

#define TILE_WIDTH 16

// Task 1 - simple matrix multiplication
__global__ void matrix_multiply_simple(float *ma, float *mb, float *mc, size_t width)
{
	//TODO: calculate the row & column index of the element
	int row = threadIdx.y;
	int col = threadIdx.x;

	//TODO: do dot product between row of ma and column of mb
	int idx = row * width + col;

	if(row<width && col <width) {
   float product_val = 0;
   for(int k=0;k<width;k++) {
      product_val += ma[row*width+k]*mb[k*width+col];
   }
   mc[idx] = product_val;
}
	

	//TODO: write result in mc
}

// Task 2 - optimized matrix multiplication
__global__ void matrix_multiply(float *ma, float *mb, float *mc, size_t width)
{
	int tx = threadIdx.x, ty = threadIdx.y;
	int bx = blockIdx.x,  by = blockIdx.y;
	int row = threadIdx.y;
	int col = threadIdx.x;
	int idx = row * width+ col;

	//TODO: allocate 2D tiles in __shared__ memory

	//TODO: calculate the row & column index of the element
	__shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];


	float result = 0;

	// loop over the tiles of the input
	for(int k = 0; k< width/TILE_WIDTH; ++k) {
	
		//TODO: load tiles into __shared__ memory allocated before
		
		//TODO:
		// wait until all data is loaded before allowing
		// any thread in this block to continue

		//TODO: do dot product between row of tile from ma and column of tile from mb

		//TODO:
		// wait until all data is loaded before allowing
		// any thread in this block to continue



		  if (k*TILE_WIDTH + threadIdx.x < width && row < width)
             As[threadIdx.y][threadIdx.x] = ma[row*width + k*TILE_WIDTH + threadIdx.x];
         else
             As[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_WIDTH + threadIdx.y < width && col < width)
             Bs[threadIdx.y][threadIdx.x] = mb[(k*TILE_WIDTH + threadIdx.y)*width + col];
         else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_WIDTH; ++n)
             result += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
	}

	//TODO: write result in mc
	mc[idx]=result;
}

int main(void)
{
	// create a large workload so we can easily measure the
	// performance difference of both implementations

	// note that n measures the width of the matrix, not the number of total elements
	const size_t n = 1<<10;
	const dim3 block_size(TILE_WIDTH,TILE_WIDTH);
	const dim3 num_blocks(n / block_size.x, n / block_size.y);

	// generate random input on the host
	std::vector<float> host_a(n*n), host_b(n*n), host_c(n*n);
	for(int i = 0; i < n*n; ++i) {
		host_a[i] = static_cast<float>(rand()) / RAND_MAX;
		host_b[i] = static_cast<float>(rand()) / RAND_MAX;
	}

	// allocate storage for the device
	float *device_a = 0, *device_b = 0, *device_c = 0;
	hipMalloc((void**)&device_a, sizeof(float) * n * n);
	hipMalloc((void**)&device_b, sizeof(float) * n * n);
	hipMalloc((void**)&device_c, sizeof(float) * n * n);

	// copy input to the device
	hipMemcpy(device_a, &host_a[0], sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(device_b, &host_b[0], sizeof(float) * n * n, hipMemcpyHostToDevice);

	//Task 3 - measure the time spent in the kernel for simple and optimized implementation
	
	//TODO: create CUDA events for measuring kernel time
	hipEvent_t launch_begin, launch_end;
	hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

	// time many kernel launches and take the average time
	const size_t num_launches = 100;
	float average_simple_time = 0;
	std::cout << "Timing simple implementation...";
	
	for(int i = 0; i < num_launches; ++i) {
		//TODO: record CUDA event before and after the kernel launch
		hipEventRecord(launch_begin);
		matrix_multiply_simple<<<num_blocks,block_size>>>(device_a, device_b, device_c, n);
		hipEventRecord(launch_end);
    	hipEventSynchronize(launch_end);

		//TODO: Wait for launch_end event to complete

		//TODO: measure the time spent in the kernel
		float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);
    float seconds = time / pow((float) 10, 3);

		average_simple_time += time;
	}
	
	average_simple_time /= num_launches;
	std::cout << " done." << std::endl;

	//now time the optimized kernel

	// time many kernel launches and take the average time
	float average_optimized_time = 0;
	std::cout << "Timing optimized implementation...";
	for(int i = 0; i < num_launches; ++i) {
		//TODO: record CUDA event before and after the kernel launch
		hipEventRecord(launch_begin);
		matrix_multiply<<<num_blocks,block_size>>>(device_a, device_b, device_c, n);

		//TODO: Wait for launch_end event to complete
		hipEventRecord(launch_end);
    	hipEventSynchronize(launch_end);
		
		//TODO: measure the time spent in the kernel

		float time = 0;

    hipEventElapsedTime(&time, launch_begin, launch_end);
    float seconds = time / pow((float) 10, 3);
		average_optimized_time += time;
	}
	average_optimized_time /= num_launches;
	std::cout << " done." << std::endl;

	// report the effective throughput of each kernel in GFLOPS
	// the effective throughput is measured as the number of floating point operations performed per second:
	// (one mul + one add) * N^3
	float simple_throughput = static_cast<float>(2 * n * n * n) / (average_simple_time / 1000.0f) / 1000000000.0f;
	float optimized_throughput = static_cast<float>(2 * n * n * n) / (average_optimized_time / 1000.0f) / 1000000000.0f;

	std::cout << "Matrix size: " << n << "x" << n << std::endl;
	std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

	std::cout << "Throughput of simple kernel: " << simple_throughput << " GFLOPS" << std::endl;
	std::cout << "Throughput of optimized kernel: " << optimized_throughput << " GFLOPS" << std::endl;
	std::cout << "Performance improvement: " << optimized_throughput / simple_throughput << "x" << std::endl;
	std::cout << std::endl;

	//TODO: destroy the CUDA events

	// deallocate device memory
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

	return 0;
}

