#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i]=a[i]+b[i];
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays

      const int num_bytes = N * sizeof(float);
      host_array_a = (float *) malloc(num_bytes);
    host_array_b = (float *) malloc(num_bytes);
    host_array_c = (float *) malloc(num_bytes);

    // TODO 2: Allocate the device's arrays
    hipMalloc((void **) &device_array_a, num_bytes);
    hipMalloc((void **) &device_array_b, num_bytes);

    hipMalloc((void **) &device_array_c, num_bytes);
    // TODO 3: Check for allocation errors

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.

fill_array_float(host_array_a,N);
fill_array_random(host_array_b,N);


    // TODO 5: Copy the host's arrays to device


    hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).

        size_t blocks_no = N / 256;

    add_arrays<<<blocks_no, 256>>>(device_array_a, device_array_b,device_array_c, N);




    // TODO 7: Copy back the results and then uncomment the checking function
    hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);

     check_task_2(host_array_a, host_array_b, host_array_c, N); 

    // TODO 8: Free the memory
   
    return 0;
}