#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Avoid accessing out of bounds elements
    printf("aici %d i %d ai",i,a[i]);
    if(i<N)
    if (a[i] %2) 
        
        a[i]=1;
    
    else 
    a[i]=0;

}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {
unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

a[i]=i / blockDim.x; 





}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {
unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

a[i]=threadIdx.x; 

}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
    

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.
    const size_t block_size = 4;
    const int num_elements =4 *4;


    int *host_array=0;
    int *device_array_a = 0;


    const int num_bytes = num_elements * sizeof(int);
    hipMalloc((void **) &device_array_a, num_bytes);
    host_array = (int *) malloc(num_bytes);



for (int i = 0; i < num_elements; ++i) {
        host_array[i] =  i;
    }
size_t blocks_no = num_elements / block_size;


  hipMemcpy(device_array_a, host_array, num_bytes, hipMemcpyHostToDevice);



    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //
    // Uncomment the line below to check your results

kernel_parity_id<<<blocks_no, block_size>>>(device_array_a ,num_elements );
hipMemcpy(host_array, device_array_a, num_bytes, hipMemcpyDeviceToHost);
    check_task_1(3, host_array); 

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results





kernel_block_id<<<blocks_no, block_size>>>(device_array_a ,num_elements );
hipMemcpy(host_array, device_array_a, num_bytes, hipMemcpyDeviceToHost);
    check_task_1(4, host_array); 

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results

kernel_thread_id<<<blocks_no, block_size>>>(device_array_a ,num_elements );
hipMemcpy(host_array, device_array_a, num_bytes, hipMemcpyDeviceToHost);
    check_task_1(5, host_array); 

    // TODO 6: Free the memory
    free(host_array);
    
    hipFree(device_array_a);
    
    return 0;
}